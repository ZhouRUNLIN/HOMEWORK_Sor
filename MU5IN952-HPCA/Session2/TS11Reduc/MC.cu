#include "hip/hip_runtime.h"
#include <stdio.h>


// Question 1
// a)
__global__ void biggest_k1a(int *In, int *Out, int N) {
  int i;
  *Out = In[0];

  for (i = 0; i < N; i++) {
    *Out = max(*Out, In[i]);
  }
}

// b)
__global__ void biggest_k1b(int *In, int *Out, int N) {
  int i;
  __shared__ int M[2];
  M[threadIdx.x] = In[threadIdx.x];

  for (i = threadIdx.x + 2; i < N; i += 2) {
    M[threadIdx.x] = max(M[threadIdx.x], In[i]);
  }
  
  if (threadIdx.x == 0) {
    *Out = max(M[0], M[1]);
  }
}

// c)
__global__ void biggest_k1c(int *In, int *Out, int N) {
  int i;
  __shared__ int M[4];
  M[threadIdx.x] = In[threadIdx.x];

  for (i = threadIdx.x + 4; i < N; i += 4) {
    M[threadIdx.x] = max(M[threadIdx.x], In[i]);
  }
  
  if (threadIdx.x < 2) {
    M[threadIdx.x] = max(M[threadIdx.x], M[threadIdx.x + 2]);
  }
  if (threadIdx.x == 0) {
    *Out = max(M[0], M[1]);
  }
}

// d)
__global__ void biggest_k1d(int *In, int *Out, int N) {
  int i;
  __shared__ int M[64];
  M[threadIdx.x] = max(In[threadIdx.x], In[threadIdx.x + 64]);
  __synrthreads();

  i = 32;
  while (i > 1) {
    if (threadIdx.x < i) {
      M[threadIdx.x] = max(M[threadIdx.x], M[threadIdx.x + i])
    }
    i /= 2;
  }
  for (i = threadIdx.x + 4; i < N; i += 4) {
    M[threadIdx.x] = max(M[threadIdx.x], In[i]);
  }

  if (threadIdx.x == 0) {
    *Out = max(M[0], M[1]);
  }
}

// Question 3
__global__ void biggest_k3(int *In, int *Out, int N) {
  int i;
  __shared__ int M[64];
  M[threadIdx.x] = In[threadIdx.x];

  for (i = threadIdx.x + 64; i < N; i += 64) {
    M[threadIdx.x] = max(M[threadIdx.x], M[threadIdx.x + 64])
  }
  __synrthreads();

  i = 32;
  while (i > 0) {
    if (threadIdx.x < i) {
      M[threadIdx.x] = max(M[threadIdx.x], M[threadIdx.x + i])
    }
    i /= 2;
  }
  for (i = threadIdx.x + 4; i < N; i += 4) {
    M[threadIdx.x] = max(M[threadIdx.x], In[i]);
  }

  if (threadIdx.x == 0) {
    *Out = max(M[0], M[1]);
  }
}

// Question 4
// a)
__global__ void biggest_k4a(int *In, int *Out, int N) {
  int i;
  int idx = threadIdx.x + blockIdx.x*blockDim.x
  __shared__ int M[64];
  M[threadIdx.x] = max(In[idx], In[idx + 128]);
  __synrthreads();

  i = 32;
  while (i > 0) { // i > 1
    if (threadIdx.x < i) {
      M[threadIdx.x] = max(M[threadIdx.x], M[threadIdx.x + i])
    }
    i /= 2;
  }

  if (threadIdx.x == 0) {
    atomicMax(Out, M[0]); // max(M[0], M[1])
  }
}

// b)
__global__ void biggest_k4b(int *In, int *Out, int N) {
  int i;
  int idx = threadIdx.x + blockIdx.x*blockDim.x
  __shared__ int M[64];

  if (idx < N) {
    M[threadIdx.x] = In[idx];
  } else {
    M[threadIdx.x] = In[idx + 128];
  }
  __synrthreads();

  i = blockDim.x / 2;
  while (i > 0) {
    if (threadIdx.x < i) {
      M[threadIdx.x] = max(M[threadIdx.x], M[threadIdx.x + i])
    }
    i /= 2;
    __synrthreads();
  }

  if (threadIdx.x == 0) {
    atomicMax(Out, M[0]);
  }
}