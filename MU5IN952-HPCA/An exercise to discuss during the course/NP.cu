/**************************************************************
This code is a part of a course on cuda taught by the author: 
Lokman A. Abbas-Turki

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define NB 2048
#define NTPB 1024

__device__ float Glob[7*NB*NTPB];	// Global variable solution
__constant__ float Cst[7];		// Constant variable solution

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {

	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}


// Has to be defined in the compilation in order to get the correct value of the macros
// __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


/*One-Dimensional Normal Law. Cumulative distribution function. */
float NP(float x)
{
  float p = 0.2316419f;
  float b1= 0.3193815f;
  float b2= -0.3565638f;
  float b3= 1.781478f;
  float b4= -1.821256f;
  float b5= 1.330274f;
  float one_over_twopi= 0.3989423f;  
  float t;

  if(x >= 0.0f){
      t = 1.0f / ( 1.0f + p * x );
      return (1.0f - one_over_twopi * expf( -x * x / 2.0f ) * t * 
			 ( t *( t * ( t * ( t * b5 + b4 ) + b3 ) + b2 ) + b1 ));
  }else{/* x < 0 */
      t = 1.0f / ( 1.0f - p * x );
      return ( one_over_twopi * expf( -x * x / 2.0f ) * t * 
			 ( t *( t * ( t * ( t * b5 + b4 ) + b3 ) + b2 ) + b1 ));
  }
}


// Shared based solution
__global__ void  NP_SHA_k(float* x)
{
	/*************************************************************

	Once requested, replace this comment by the appropriate code

	*************************************************************/
}


// Constant memory based solution
__global__ void  NP_CST_k(float* x)
{
	/*************************************************************

	Once requested, replace this comment by the appropriate code

	*************************************************************/
}


// Global memory based solution with read only caching
__global__ void  NP_GLOreadOnly_k(float* x)
{
	/*************************************************************

	Once requested, replace this comment by the appropriate code

	*************************************************************/
}

// Global memory based solution
__global__ void  NP_GLO_k(float *x)
{
  int idx = threadIdx.x + blockIdx.x*blockDim.x;
  Glob[idx] = 0.2316419f;
  Glob[idx + NB * NTPB] = 0.3193815f;
  Glob[idx + NB * NTPB * 2] = -0.3565638f;
  Glob[idx + NB * NTPB * 3] = 1.781478f;
  Glob[idx + NB * NTPB * 4] = -1.821256f;
  Glob[idx + NB * NTPB * 5] = 1.330274f;
  Glob[idx + NB * NTPB * 6] = 0.3989423f;  
  float t, X;

  X = x[idx];

  if(X >= 0.0f){
      t = 1.0f / ( 1.0f + Glob[idx] * X );
      x[idx] = (1.0f - Glob[idx+6* NB * NTPB] * expf( -X * X / 2.0f ) * t *
			   ( t *( t * ( t * ( t * Glob[idx+5* NB * NTPB] + Glob[idx+4* NB * NTPB] ) +
			   Glob[idx+3* NB * NTPB] ) + Glob[idx+2* NB * NTPB] ) + Glob[idx+ NB * NTPB] ));
  }else{/* X < 0 */
      t = 1.0f / ( 1.0f - Glob[idx] * X );
      x[idx] = ( Glob[idx+6* NB * NTPB] * expf( -X * X / 2.0f ) * t *
			   ( t *( t * ( t * ( t * Glob[idx+5* NB * NTPB] + Glob[idx+4* NB * NTPB] ) +
			   Glob[idx+3* NB * NTPB] ) + Glob[idx+2* NB * NTPB] ) + Glob[idx+ NB * NTPB] ));

  }
}

// The wrapper
void NP_GPU(float *a, float *b, int flag, float *TimerAdd){

	float *aGPU;

	hipEvent_t start, stop;				// GPU timer instructions
	testCUDA(hipEventCreate(&start));		// GPU timer instructions
	testCUDA(hipEventCreate(&stop));		// GPU timer instructions

	testCUDA(hipMalloc(&aGPU, NB*NTPB*sizeof(float)));
	testCUDA(hipMemcpy(aGPU, a, NB*NTPB*sizeof(float), hipMemcpyHostToDevice));

	testCUDA(hipEventRecord(start,0));		// GPU timer instructions
	// Launching the operation on the GPU
	if (flag == 0)	NP_GLO_k<<<NB, NTPB>>>(aGPU);
	if (flag == 1)	NP_GLOreadOnly_k<<<NB, NTPB>>>(aGPU);
	if (flag == 2)	NP_CST_k<<<NB, NTPB>>>(aGPU);
	if (flag == 3)	NP_SHA_k<<<NB, NTPB>>>(aGPU);

	testCUDA(hipEventRecord(stop,0));		// GPU timer instructions
	testCUDA(hipEventSynchronize(stop));	// GPU timer instructions
	testCUDA(hipEventElapsedTime(TimerAdd,	// GPU timer instructions
			 start, stop));					// GPU timer instructions

	// Copying the value from one ProcUnit to the other ProcUnit
	testCUDA(hipMemcpy(b, aGPU, NB*NTPB*sizeof(float), hipMemcpyDeviceToHost));

	// Freeing the GPU memory
	testCUDA(hipFree(aGPU));
	testCUDA(hipEventDestroy(start));		// GPU timer instructions
	testCUDA(hipEventDestroy(stop));		// GPU timer instructions
}


int main (void){

	// Variables definition
	float *a, *b;
	float TimerAdd;
	int i;
	
	// Length for the size of arrays
	int length = NB*NTPB;

	// Memory allocation of arrays 
	a = (float*)malloc(length*sizeof(float));
	b = (float*)malloc(length*sizeof(float));

	// Setting values
	for(i=0; i<length; i++){
		a[i] = (float)(i-length/2.0f)/length;
	}


	// Warming the GPU
	NP_GPU(a, b, 0, &TimerAdd);

	// Executing the different options 
	NP_GPU(a, b, 0, &TimerAdd);
	printf("Execution time using large global: %f ms\n", TimerAdd);
	printf(" %f is equal to %f \n", b[5], NP(a[5])); // Check the result 
	NP_GPU(a, b, 1, &TimerAdd);
	printf("Execution time using large read-only global: %f ms\n", TimerAdd);
	printf(" %f is equal to %f \n", b[25], NP(a[25])); // Check the result
	NP_GPU(a, b, 2, &TimerAdd);
	printf("Execution time using constant memory: %f ms\n", TimerAdd);
	printf(" %f is equal to %f \n", b[250], NP(a[250])); // Check the result 
	NP_GPU(a, b, 3, &TimerAdd);
	printf("Execution time using shared memory: %f ms\n", TimerAdd);
	printf(" %f is equal to %f \n", b[577], NP(a[577])); // Check the result 

	// Freeing the memory
	free(a);
	free(b);

	return 0;
}