/**************************************************************
This code is a part of a course on cuda taught by the author: 
Lokman A. Abbas-Turki

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

// add kernel
__global__ void add_k(int *A, int *B, int N){

	//for (int i = 0; i < 50; i++) { // or 1000
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < N) {
			B[idx] += A[idx];
		}
	//}
}

// subtract kernel
__global__ void sub_k(int *A, int *B, int N){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx<N){
		B[idx] -= A[idx];
	}
}


/////////////////////////////////////////////////////////////////////////
// To show the difference between transferring 
// everything once vs. using smaller frames 
/////////////////////////////////////////////////////////////////////////
void withoutStream (int *aCPU, int *bCPU, int size, int NBS){

	int Qsize = 128;
	int F = size/Qsize;
	size_t L = F*sizeof(int);
	int *a, *b, *cCPU; 
	float TimeVar;
	hipEvent_t start, stop;

	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));
	hipHostAlloc(&cCPU, size*sizeof(int), hipHostMallocDefault);
	testCUDA(hipMalloc(&a,size*sizeof(int)));
	testCUDA(hipMalloc(&b,size*sizeof(int)));
	testCUDA(hipEventRecord(start,0));

	if(NBS<0){
	  testCUDA(hipMemcpy(a,aCPU,size*sizeof(int),hipMemcpyHostToDevice));
	  testCUDA(hipMemcpy(b,bCPU,size*sizeof(int),hipMemcpyHostToDevice));
	  add_k<<<(size+1023)/1024,1024>>>(a,b,size);
	  testCUDA(hipMemcpy(cCPU,b,size*sizeof(int),hipMemcpyDeviceToHost));
	}else{
  	  for (int i=0; i<size; i+= NBS*F){
		for (int j=0; j <NBS; j++){
		  testCUDA(hipMemcpy(a+j*F+i,aCPU+j*F+i,L,hipMemcpyHostToDevice));
		  testCUDA(hipMemcpy(b+j*F+i,bCPU+j*F+i,L,hipMemcpyHostToDevice));
		}
		for (int j=0; j <NBS; j++){
		  add_k<<<(F+1023)/1024,1024>>>(a+j*F+i,b+j*F+i,F);
		}
		for (int j=0; j <NBS; j++){
		  testCUDA(hipMemcpy(cCPU+j*F+i,b+j*F+i,L,hipMemcpyDeviceToHost));
		}
	  }
	}

	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	for (int i=size-F-3; i<size-F; i++){
		printf("a[i]+b[i] = %i,  %i\n",aCPU[i]+bCPU[i],cCPU[i]);
	}
	if(NBS<0){
		printf("Processing time for doing everything once: %f ms\n", TimeVar);
	}else{
		printf("Processing time when using frames without streams: %f ms\n", TimeVar);
	}
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipFree(a));
	testCUDA(hipFree(b));
	hipHostFree(cCPU);
}


int main (void){

	//Compare using different sizes
	// Small size
	int size = 32*512*512;
	// Average size
	//int size = 128*512*512;
	// Big size
	//int size = 512 * 512 * 512;


	// Number of streams 
	int NBS = 16;

	int *a, *b;

	hipHostAlloc(&a, size*sizeof(int), hipHostMallocDefault);
	hipHostAlloc(&b, size*sizeof(int), hipHostMallocDefault);

	for (int i=0; i<size; i++){
		a[i] = i;
		b[i] = 2*i;
	}

	// Warming up the GPU
	withoutStream (a, b, size, -1);

	// Only one transfer
	withoutStream (a, b, size, -1);
	// Transfer frames without streams: Processing them NBS by NBS frames
	withoutStream (a, b, size, NBS);

	// Transfer frames using NBS streams: Processing them NBS by NBS frames
	//withStream (a, b, size, NBS);

	hipHostFree(a);
	hipHostFree(b);
}