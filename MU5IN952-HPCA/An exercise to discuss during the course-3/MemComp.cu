/**************************************************************
This code compares standard CPU allocation with the locked one.
It also checks the effeciency of mapping the CPU memory 

This code is a part of a course on cuda taught by the author: 
Lokman A. Abbas-Turki

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

// This kernel is needed to compare the mapped memory to other memories
__global__ void test_kernel(int *Tab, int size, int i){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx<size){
		Tab[idx] = i;
	}
}

float malloc_trans(int size, int NbT, bool flag) {

	int *a, *aGPU;
	float TimeVar = 0.0f;
	hipEvent_t start, stop;
	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));

	a = (int*)malloc(size*sizeof(int));
	testCUDA(hipMalloc(&aGPU,size*sizeof(int)));

	testCUDA(hipEventRecord(start,0));

	for (int i=0; i<NbT; i++) {
		if (flag){
			testCUDA(hipMemcpy(aGPU, a, size*sizeof(int),	hipMemcpyHostToDevice)); 
			// test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
		}else{
			// test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
			testCUDA(hipMemcpy(a, aGPU, size*sizeof(int),	hipMemcpyDeviceToHost));
		}
	}

	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipFree(aGPU));
	free(a);	
	return TimeVar;
}

float hostAlloc_trans(int size, int NbT, bool flag) {
	int *a, *aGPU;
	float TimeVar = 0.0f;
	hipEvent_t start, stop;
	testCUDA(hipEventCreate(&start));
	testCUDA(hipEventCreate(&stop));

	a = (int*)malloc(size*sizeof(int));
	testCUDA(hipMalloc(&aGPU,size*sizeof(int)));

	testCUDA(hipEventRecord(start,0));

	for (int i=0; i<NbT; i++) {
		if (flag){
			testCUDA(hipMemcpy(aGPU, a, size*sizeof(int),	hipMemcpyHostToDevice)); 
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
		}else{
			test_kernel<<<(size+127)/128,128>>>(aGPU,size,i);//Comparison with mapped
			testCUDA(hipMemcpy(a, aGPU, size*sizeof(int),	hipMemcpyDeviceToHost));
		}
	}

	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipFree(aGPU));
	free(a);	
	return TimeVar;
}

float mappedAlloc_trans(int size, int NbT) {

	float TimeVar = 0.0f;
	/*************************************************************

	Once requested, replace this comment by the appropriate code

	*************************************************************/
	return TimeVar;
}

int main (void){

	int size = 1024*1024;
	int NbT = 100;
	float TimeVar;

	
	TimeVar = malloc_trans(size, NbT, true);
	printf("Processing time when using malloc CPU2GPU: %f s\n", 
		   0.001f*TimeVar);
	TimeVar = malloc_trans(size, NbT, false);
	printf("Processing time when using malloc GPU2CPU: %f s\n", 
		   0.001f*TimeVar);

	TimeVar = hostAlloc_trans(size, NbT, true);
	printf("Processing time when using hipHostAlloc CPU2GPU: %f s\n", 
		   0.001f*TimeVar);
	TimeVar = hostAlloc_trans(size, NbT, false);
	printf("Processing time when using hipHostAlloc GPU2CPU: %f s\n", 
		   0.001f*TimeVar);

	TimeVar = mappedAlloc_trans(size, NbT);
	printf("Processing time for mapped memory: %f s\n", 
		   0.001f*TimeVar);
	return 0;
}

