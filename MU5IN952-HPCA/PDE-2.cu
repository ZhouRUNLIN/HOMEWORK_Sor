#include "hip/hip_runtime.h"
/**************************************************************
This code is a part of a course on cuda taught by the author:
Lokman A. Abbas-Turki

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/

#include <stdio.h>
#include <math.h>

#define EPS 0.0000001f
#define NTPB 256
#define NB 64
#define r 0.1f

typedef float MyTab[NB][NTPB];

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {

	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

/*************************************************************************/
/*                   Black-Sholes Formula                                */
/*************************************************************************/
/*One-Dimensional Normal Law. Cumulative distribution function. */
double NP(double x){
  const double p= 0.2316419;
  const double b1= 0.319381530;
  const double b2= -0.356563782;
  const double b3= 1.781477937;
  const double b4= -1.821255978;
  const double b5= 1.330274429;
  const double one_over_twopi= 0.39894228;  
  double t;

  if(x >= 0.0){
	t = 1.0 / ( 1.0 + p * x );
    return (1.0 - one_over_twopi * exp( -x * x / 2.0 ) * t * ( t *( t * 
		   ( t * ( t * b5 + b4 ) + b3 ) + b2 ) + b1 ));
  }else{/* x < 0 */
    t = 1.0 / ( 1.0 - p * x );
    return ( one_over_twopi * exp( -x * x / 2.0 ) * t * ( t *( t * ( t * 
		   ( t * b5 + b4 ) + b3 ) + b2 ) + b1 ));
  }
}


// Parallel cyclic reduction for implicit part
__device__ void PCR_d(float* sa, float* sd, float* sc,
	float* sy, int* sl, int n) {

	int i, lL, d, tL, tR;
	float aL, dL, cL, yL;
	float aLp, dLp, cLp, yLp;

	d = (n / 2 + (n % 2)) * (threadIdx.x % 2) + (int)threadIdx.x / 2;

	tL = threadIdx.x - 1;
	if (tL < 0) tL = 0;
	tR = threadIdx.x + 1;
	if (tR >= n) tR = 0;

	for (i = 0; i < (int)(logf((float)n) / logf(2.0f)) + 1; i++) {
		lL = (int)sl[threadIdx.x];

		aL = sa[threadIdx.x];
		dL = sd[threadIdx.x];
		cL = sc[threadIdx.x];
		yL = sy[threadIdx.x];

		dLp = sd[tL];
		cLp = sc[tL];

		if (fabsf(aL) > EPS) {
			aLp = sa[tL];
			yLp = sy[tL];
			dL -= aL * cL / dLp;
			yL -= aL * yLp / dLp;
			aL = -aL * aLp / dLp;
			cL = -cLp * cL / dLp;
		}

		cLp = sc[tR];
		if (fabsf(cLp) > EPS) {
			aLp = sa[tR];
			dLp = sd[tR];
			yLp = sy[tR];
			dL -= cLp * aLp / dLp;
			yL -= cLp * yLp / dLp;
		}
		__syncthreads();

		if (i < (int)(logf((float)n) / logf(2.0f))) {
			sa[d] = aL;
			sd[d] = dL;
			sc[d] = cL;
			sy[d] = yL;
			sl[d] = (int)lL;
			__syncthreads();
		}
	}

	sy[(int)sl[threadIdx.x]] = yL / dL;
}


/////////////////////////////////////////////////////////////////////////////
// A bad solution that makes a lot of accesses to the global memory
/////////////////////////////////////////////////////////////////////////////
__global__ void PDE_diff_k1 (float dt, float dx, float dsig, float pmin, 
							 float pmax, float sigmin, MyTab *pt_GPU){
	int u = threadIdx.x+1;
	int m = threadIdx.x;
	int d = threadIdx.x-1;
    float sig = sigmin + dsig*blockIdx.x;
	float mu = r - 0.5f*sig*sig;
	float pu = 0.5f*(sig*sig*dt/(dx*dx) + mu*dt/dx);
	float pm = 1.0f - sig*sig*dt/(dx*dx);
	float pd = 0.5f*(sig*sig*dt/(dx*dx) - mu*dt/dx);

	__shared__ float A1[NTPB];
	__shared__ float A2[NTPB];

	A1[threadIdx.x] = pt_GPU[0][blockIdx.x][threadIdx.x];
	__syncthreads();

	if(threadIdx.x==0){
	  A2[threadIdx.x] = pmin;
	}else{
	  if(threadIdx.x==NTPB-1){
	    A2[threadIdx.x] = pmax;
	  }else{
	    A2[threadIdx.x] = pu*A1[u] + pm*A1[m] + pd*A1[d];	  
	  }
	}
	__syncthreads();

	pt_GPU[0][blockIdx.x][threadIdx.x] = A2[threadIdx.x];
}


/////////////////////////////////////////////////////////////////////////////
// The shared dynamic allocation version of the optimal explicit solution
/////////////////////////////////////////////////////////////////////////////
__global__ void PDE_diff_k2(float dt, float dx, float dsig, float pmin,
	float pmax, float sigmin, int N, MyTab* pt_GPU) {
	int i;
	int u = threadIdx.x + 1;
	int m = threadIdx.x;
	int d = threadIdx.x - 1;
	float sig = sigmin + dsig * blockIdx.x;
	float mu = r - 0.5f * sig * sig;
	float pu = 0.5f * (sig * sig * dt / (dx * dx) + mu * dt / dx);
	float pm = 1.0f - sig * sig * dt / (dx * dx);
	float pd = 0.5f * (sig * sig * dt / (dx * dx) - mu * dt / dx);

	extern __shared__ float A[];

	float* A1 = A;
	float* A2 = A + NTPB;

	A1[threadIdx.x] = pt_GPU[0][blockIdx.x][threadIdx.x];
	__syncthreads();

	for (i = 1; i <= N; i++) {
		if ((i % 2)) {
			if (threadIdx.x == 0) {
				A2[threadIdx.x] = pmin;
			}
			else {
				if (threadIdx.x == NTPB - 1) {
					A2[threadIdx.x] = pmax;
				}
				else {
					A2[threadIdx.x] = pu * A1[u] + pm * A1[m] + pd * A1[d];
				}
			}
			__syncthreads();
		}
		else {
			if (threadIdx.x == 0) {
				A1[threadIdx.x] = pmin;
			}
			else {
				if (threadIdx.x == NTPB - 1) {
					A1[threadIdx.x] = pmax;
				}
				else {
					A1[threadIdx.x] = pu * A2[u] + pm * A2[m] + pd * A2[d];
				}
			}
			__syncthreads();
		}
	}

	if (N % 2) {
		pt_GPU[0][blockIdx.x][threadIdx.x] = A2[threadIdx.x];
	}
	else {
		pt_GPU[0][blockIdx.x][threadIdx.x] = A1[threadIdx.x];
	}
}

/////////////////////////////////////////////////////////////////////////////
// Implicit solution
/////////////////////////////////////////////////////////////////////////////
__global__ void PDE_diff_k3(float dt, float dx, float dsig, float pmin,
	float pmax, float sigmin, int N, MyTab* pt_GPU) {
	int i;
	float sig = sigmin + dsig * blockIdx.x;
	float mu = r - 0.5f * sig * sig;
	float qu = -0.5f * (sig * sig * dt / (dx * dx) + mu * dt / dx);
	float qm = 1.0f + sig * sig * dt / (dx * dx);
	float qd = -0.5f * (sig * sig * dt / (dx * dx) - mu * dt / dx);

	extern __shared__ float A[];

	float* sa = A;
	float* sd = sa + NTPB;
	float* sc = sd + NTPB;
	float* sy = sc + NTPB;
	int* sl = (int*)sy + NTPB;

	sy[threadIdx.x] = pt_GPU[0][blockIdx.x][threadIdx.x];
	__syncthreads();

	for (i = 1; i <= N; i++) {
		sd[threadIdx.x] = qm;
		if (threadIdx.x < NTPB - 1) {
			sc[threadIdx.x + 1] = qu;
		}
		if (threadIdx.x > 0) {
			sa[threadIdx.x] = qd;
		}
		if (threadIdx.x == 0) {
			sa[0] = 0.f;
			sc[0] = 0.f;
		}
		sl[threadIdx.x] = threadIdx.x;

		__syncthreads();
		PCR_d(sa, sd, sc, sy, sl, NTPB);
		__syncthreads();

		if (threadIdx.x == 0) {
			sy[0] = pmin;
			sy[NTPB - 1] = pmax;
		}
	}

	pt_GPU[0][blockIdx.x][threadIdx.x] = sy[threadIdx.x];
}

/////////////////////////////////////////////////////////////////////////////
// Crank-Nicolson solution
/////////////////////////////////////////////////////////////////////////////
__global__ void PDE_diff_k4 (float dt, float dx, float dsig, float pmin, 
							 float pmax, float sigmin, int N, MyTab *pt_GPU){
    int i;
	int u = threadIdx.x + 1;
	int m = threadIdx.x;
	int d = threadIdx.x - 1;
    float sig = sigmin + dsig*blockIdx.x;
	float mu = r - 0.5f*sig*sig;
	float pu = 0.25f*(sig*sig*dt/(dx*dx) + mu*dt/dx);
	float pm = 1.0f - 0.5*sig*sig*dt/(dx*dx);
	float pd = 0.25f*(sig*sig*dt/(dx*dx) - mu*dt/dx);
	float qu = -0.25f * (sig * sig * dt / (dx * dx) + mu * dt / dx);
	float qm = 1.0f + 0.5*sig * sig * dt / (dx * dx);
	float qd = -0.25f * (sig * sig * dt / (dx * dx) - mu * dt / dx);

	extern __shared__ float A[];

	float* sa = A;
	float* sd = sa + NTPB;
	float* sc = sd + NTPB;
	float* sy = sc + NTPB;
	int* sl = (int*)sy + 2*NTPB;

	sy[m] = pt_GPU[0][blockIdx.x][m];
	__syncthreads();

	for (i = 1; i <= N; i++) {
		if (m == 0) {
			sy[NTPB*(i%2) + m] = pmin;
		}
		else {
			if (m == NTPB - 1) {
				sy[NTPB*(i%2) + m] = pmax;
			}
			else {
				sy[NTPB*(i%2) + m] = pu*sy[NTPB * ((i+1) % 2) + u] +
									 pm*sy[NTPB * ((i+1) % 2) + m] +
									 pd*sy[NTPB * ((i+1) % 2) + d];
			}
		}
		sd[m] = qm;
		if (m < NTPB - 1) {
			sc[m + 1] = qu;
		}
		if (m > 0) {
			sa[m] = qd;
		}
		if (m == 0) {
			sa[0] = 0.f;
			sc[0] = 0.f;
		}
		sl[m] = m;

		__syncthreads();
		PCR_d(sa, sd, sc, sy + NTPB * (i % 2), sl, NTPB);
		__syncthreads();

		if (m == 0) {
			sy[NTPB * (i % 2)] = pmin;
			sy[NTPB * (i % 2) + NTPB - 1] = pmax;
		}
		__syncthreads();
	}

	pt_GPU[0][blockIdx.x][m] = sy[m+ NTPB*(N % 2)];
}



// Wrapper 
void PDE_diff (float dt, float dx, float dsig, float pmin, float pmax, 
			   float sigmin, int N, MyTab* CPUTab){

	float TimeExec;									// GPU timer instructions
	hipEvent_t start, stop;						// GPU timer instructions
	testCUDA(hipEventCreate(&start));				// GPU timer instructions
	testCUDA(hipEventCreate(&stop));				// GPU timer instructions
	testCUDA(hipEventRecord(start,0));				// GPU timer instructions

	MyTab *GPUTab;
	testCUDA(hipMalloc(&GPUTab, sizeof(MyTab)));
	
	testCUDA(hipMemcpy(GPUTab, CPUTab, sizeof(MyTab), hipMemcpyHostToDevice));
	// Accessing 2*N times to the global memory
	/*for(int i=0; i<N; i++){
	   PDE_diff_k1<<<NB,NTPB>>>(dt, dx, dsig, pmin, pmax, sigmin, GPUTab);
	}*/
	// Accessing only twice to the global memory
	//PDE_diff_k2<<<NB,NTPB,2*NTPB*sizeof(float)>>>(dt, dx, dsig, pmin, pmax, 
	//											  sigmin, N, GPUTab);
	//PDE_diff_k3<<<NB, NTPB, 5*NTPB*sizeof(float)>>>(dt, dx, dsig, pmin, pmax,
	//	sigmin, N, GPUTab);
	PDE_diff_k4<<<NB, NTPB, 6*NTPB*sizeof(float)>>>(dt, dx, dsig, pmin, pmax,
		sigmin, N, GPUTab);

	testCUDA(hipMemcpy(CPUTab, GPUTab, sizeof(MyTab), hipMemcpyDeviceToHost));

	testCUDA(hipEventRecord(stop,0));				// GPU timer instructions
	testCUDA(hipEventSynchronize(stop));			// GPU timer instructions
	testCUDA(hipEventElapsedTime(&TimeExec,		// GPU timer instructions
			 start, stop));							// GPU timer instructions
	testCUDA(hipEventDestroy(start));				// GPU timer instructions
	testCUDA(hipEventDestroy(stop));				// GPU timer instructions

	printf("GPU time execution for PDE diffusion: %f ms\n", TimeExec);

	testCUDA(hipFree(GPUTab));	
}

///////////////////////////////////////////////////////////////////////////
// main function for a put option f(x) = max(0,K-x)
///////////////////////////////////////////////////////////////////////////
int main(void){

	float K = 100.0f;
	float T = 1.0f;
	int N = 10000;
	float dt = (float)T/N;
	float xmin = log(0.5f*K);
	float xmax = log(2.0f*K);
	float dx = (xmax-xmin)/NTPB;
	float pmin = 0.5f*K;
	float pmax = 0.0f;
	float sigmin = 0.1f;
	float sigmax = 0.5f;
	float dsig = (sigmax-sigmin)/NB;
	

	MyTab *pt_CPU;
	testCUDA(hipHostAlloc(&pt_CPU, sizeof(MyTab), hipHostMallocDefault));
	for(int i=0; i<NB; i++){
	   for(int j=0; j<NTPB; j++){
	      pt_CPU[0][i][j] = max(0.0, K-exp(xmin + dx*j));	
	   }	
	}

	PDE_diff(dt, dx, dsig, pmin, pmax, sigmin, N, pt_CPU);

    // S0 = 100 , sigma = 0.2
	printf(" %f, compare with %f\n",exp(-r*T)*pt_CPU[0][16][128],
		   K*(exp(-r*T)*NP(-(r-0.5*0.2*0.2)*sqrt(T)/0.2)-
		   NP(-(r+0.5*0.2*0.2)*sqrt(T)/0.2)));
	// S0 = 100 , sigma = 0.3
	printf(" %f, compare with %f\n",exp(-r*T)*pt_CPU[0][32][128],
		   K*(exp(-r*T)*NP(-(r-0.5*0.3*0.3)*sqrt(T)/0.3)-
		   NP(-(r+0.5*0.3*0.3)*sqrt(T)/0.3)));
	// S0 = 141.4214 , sigma = 0.3
	printf(" %f, compare with %f\n",exp(-r*T)*pt_CPU[0][32][192],
		   K*exp(-r*T)*NP(-(log(141.4214/K)+(r-0.5*0.3*0.3)*T)/(0.3*sqrt(T)))-
		   141.4214*NP(-(log(141.4214/K)+(r+0.5*0.3*0.3)*T)/(0.3*sqrt(T))));

	testCUDA(hipHostFree(pt_CPU));	
	return 0;
}