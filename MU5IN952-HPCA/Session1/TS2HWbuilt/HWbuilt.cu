/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code
the name of the author above.
***************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

__global__ void empty_k(void) {}

__global__ void print_k(void) {

	/*************************************************************

	Once requested, replace this comment by the appropriate code

	*************************************************************/

}

int main(void) {

	empty_k <<<1, 1>>> ();

    int device = 0;
    int major = 0, minor = 0;
    size_t printf_fifo_size = 0;
    hipError_t error_id;

    // a) Use cudaDeviceGetAttribute to get the major and minor compute capability
    error_id = hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
    if (error_id != hipSuccess) {
        printf("hipDeviceGetAttribute (major) returned %d -> %s\n", (int)error_id, hipGetErrorString(error_id));
        return 1;
    }

    error_id = hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
    if (error_id != hipSuccess) {
        printf("hipDeviceGetAttribute (minor) returned %d -> %s\n", (int)error_id, hipGetErrorString(error_id));
        return 1;
    }

    printf("Device compute capability: Major = %d, Minor = %d\n", major, minor);

    // Explain why cudaDeviceGetAttribute is faster than cudaGetDeviceProperties
    printf("hipDeviceGetAttribute is faster because it directly queries specific attributes without loading all the properties of the device.\n");

    // b) Use cudaDeviceGetLimit to get the size of the FIFO buffer used for printf on the device
    error_id = hipDeviceGetLimit(&printf_fifo_size, hipLimitPrintfFifoSize);
    if (error_id != hipSuccess) {
        printf("hipDeviceGetLimit (printf FIFO size) returned %d -> %s\n", (int)error_id, hipGetErrorString(error_id));
        return 1;
    }

    printf("Size of the printf FIFO buffer: %zu bytes\n", printf_fifo_size);

    // Explanation on why we need cudaDeviceSynchronize when using printf on the device
    printf("hipDeviceSynchronize is needed after printf because device-side printf uses a FIFO buffer to transfer data to the host. hipDeviceSynchronize ensures that the kernel completes execution and all the data in the printf buffer is transferred to the host before continuing execution on the host.\n");

    return 0;
}