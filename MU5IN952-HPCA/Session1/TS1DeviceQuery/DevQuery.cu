/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code
the name of the author above.
***************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>


// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {

	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

int main() {
    int device = 0;
	int deviceCount = 0;
    // Question 1.2.1
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d -> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("No CUDA devices found!\n");
        return 1;
    }
    printf("Detected %d CUDA capable device(s).\n", deviceCount);

    // 1.2.4
    struct hipDeviceProp_t deviceProp;
    // error_id = cudaGetDeviceProperties(&deviceProp, device);

    // b)
    // error_id = cudaGetDeviceProperties(&deviceProp, 20);
    // RETURN :: cudaGetDeviceProperties returned 101 -> invalid device ordinal
    testCUDA(hipGetDeviceProperties(&deviceProp, 20));    // c)
    // There is an error in file DevQuery.cu at line 40
    
    // if (error_id != cudaSuccess) {
    //     printf("cudaGetDeviceProperties returned %d -> %s\n", (int)error_id, cudaGetErrorString(error_id));
    //     return 1;
    // }

    // a) Global memory size
    printf("Global memory size: %zu bytes\n", deviceProp.totalGlobalMem);
    printf("Largest single-precision floating-point array: %zu elements\n", deviceProp.totalGlobalMem / sizeof(float));

    // b) Maximum grid size (maxGridSize)
    // maxGridSize表示在特定设备上，每个网格维度Grid Dimension的最大尺寸。
    // CUDA网格是由块组成的三维结构，因此 maxGridSize 是一个包含三个元素的数组，分别表示在 X、Y 和 Z 方向上每个维度可以有多少个块。
    // maxGridSize[0], [1], [2]分别表示x, y, z上的三个方向上的最大块数
    printf("Max grid size: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

    // c) Maximum number of threads per block (maxThreadsDim and maxThreadsPerBlock)
    // maxThreadsDim表示每个线程块在特定设备上的每个维度的最大线程数。类似于 maxGridSize，maxThreadsDim 是一个包含三个元素的数组，分别对应于线程块的 X、Y 和 Z 维度的最大大小
    printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Max thread dimensions: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);

    // d) Warp size
    // warp是GPU调度线程的最小单位。
    // CUDA 线程是分组执行的，每个warp中的线程同时执行同一条指令，只是每个线程处理的数据可能不同
    printf("Warp size: %d threads\n", deviceProp.warpSize);

    // e) Shared memory size per block
    printf("Shared memory per block: %zu bytes\n", deviceProp.sharedMemPerBlock);

    // f) Number of registers per block
    printf("Registers per block: %d\n", deviceProp.regsPerBlock);

    // g) 2D texture size
    // 由于纹理缓存的大小非常有限，不可能将整个大尺寸的纹理放入缓存中。
    // 但是，CUDA 会利用纹理缓存来缓存局部访问的数据，从而提高性能。
    printf("Max 2D texture size: %d x %d\n", deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1]);

    // h) Number of CUDA cores
    int cudaCores = 0;
    int multiprocessorCount = deviceProp.multiProcessorCount;
    int cudaCoresPerSM = 128;  // 通常每个SM有128个CUDA核心
    cudaCores = multiprocessorCount * cudaCoresPerSM;
    printf("CUDA cores: %d\n", cudaCores);

    return 0;
}
