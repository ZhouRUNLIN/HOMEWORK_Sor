#include "hip/hip_runtime.h"
/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code
the name of the author above.
***************************************************************/

#include <stdio.h>
#include "timer.h"

// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


void addVect(int *a, int *b, int *c, int length){

	int i;

	for(i=0; i<length; i++){
		c[i] = a[i] + b[i];
	}
}

__global__ void addVect_k(int* a, int* b, int* c, int length){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	while (idx < length) {
		c[idx] = a[idx] + b[idx];
		idx += blockDim.x * gridDim.x;
	}
}


int main (void){

	// Variables definition
	int *a, *b, *c;
	int *aGPU, *bGPU, *cGPU;
	int i;
	
	// Length for the size of arrays
	int length = 1e8;
	Timer Tim;							// CPU timer instructions

	// Allocate aGPU, bGPU, cGPU on the GPU
	testCUDA(hipMalloc(&aGPU, length*sizeof(int)));
	testCUDA(hipMalloc(&bGPU, length*sizeof(int)));
	testCUDA(hipMalloc(&cGPU, length*sizeof(int)));

	// Memory allocation of arrays 
	a = (int*)malloc(length*sizeof(int));
	b = (int*)malloc(length*sizeof(int));
	c = (int*)malloc(length*sizeof(int));


	// Values initialization
	for(i=0; i<length; i++){
		a[i] = i;
		b[i] = 9*i;
	}
	
	// Transfer the values of a, b to aGPU, bGPU
	testCUDA(hipMalloc(aGPU, a, length*sizeof(int), hipMemcpyHostToDevice));
	testCUDA(hipMalloc(bGPU, b, length*sizeof(int), hipMemcpyHostToDevice));

	Tim.start();						// CPU timer instructions

	// Executing the addition 
	// addVect(a, b, c, length);
	addVect_k<<<1000, 1024>>>(aGPU, bGPU, cGPU, length);
	hipDeviceSynchronize();

	Tim.add();							// CPU timer instructions
	testCUDA(hipMalloc(c, cGPU, length*sizeof(int), hipMemcpyHostToDevice));	

	// Displaying the results to check the correctness 
	for(i=length-50; i<length-45; i++){
		printf(" ( %i ): %i\n", a[i]+b[i], c[i]);
	}

	printf("CPU Timer for the addition on the CPU of vectors: %f s\n", 
		   (float)Tim.getsum());		// CPU timer instructions

	// Freeing the memory
	free(a);
	free(b);
	free(c);

	return 0;
}