#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

// Question 1:
//
// multiBatch_k1<<<2, 512, 512*d*d*sizeof(float)>>>(A, B, d);
// multiBatch_k1<<<16, 64, 64*d*d*sizeof(float)>>>(A, B, d);
__global__ void multiBatch_k(float *A, float *B, int d) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x)*d*d; 
    int tid = threadIdx.x*d*d;
    extern __shared__ float M[];
    int i, j, k;
    float sum;

    for (i = 0; i < d*d; i++) {
        M[tid + 1] = A[idx + i];
    }

    for (i = 0; i < d; i++) {
        for (j = 0; j < d; j++) {
            sum = 0.0f;
            for (k = 0; k < d; k++) {
                sum += M[tid + i*d + k]*B[idx + k*d + j];
            }
            A[idx + i*d + j] = sum;
        }
    }
}

// Question 2:
//
// multiBatch_k2<<<64, d*d*16, 16*d*d*sizeof(float)>>>(A, B, d);
// multiBatch_k2<<<1024, d*d, d*d*sizeof(float)>>>(A, B, d);
__global__ void multiBatch_k3(float *A, float *B, int d) {
    int midx = threadIdx.x/(d*d); 
    int tid = threadIdx.x - (midx*d*d);     // threadIdx.x%(d*d)

    int ldx = tidx/d;
    int cdx = tidx - (ldx*d);

    int gidx = midx + blockIdx.x*blockDim.x/(d*d);

    extern __shared__ float M[];
    int k;
    float sum = 0.0f;

    M[threadIdx.x] = A[gidx*d*d + tidx];
    __syncthreads();

    for (k = 0; k < d; k++) {
        sum += M[midx*d*d + ldx*d + k]*B[gidx*d*d + k*d + cdx];
    }
    A[gidx*d*d + tidx] = sum;
}

// Question 3:
//
// multiBatch_k3<<<64, d*d*16, 16*d*d*sizeof(float)>>>(A, B, d);
// multiBatch_k3<<<1024, d*d, d*d*sizeof(float)>>>(A, B, d);
__global__ void multiBatch_k3(float *A, float *B, int d) {
    int midx = threadIdx.x/d; 
    int tid = threadIdx.x - (midx*d);     // threadIdx.x%d

    int gidx = midx + blockIdx.x*blockDim.x/d;

    extern __shared__ float M[];
    int i, k;
    float sum;

    for (i = 0; i < d; i++ ) {
        M[threadIdx.x] = A[gidx*d*d + i*d + tidx];
        __syncthreads();
        sum = 0.0f;
        for (k = 0; k < d; k++) {
            sum += M[midx*d + i*d + k]*B[gidx*d*d + k*d + tidx];
        }
        A[gidx*d*d + i*d + tidx] = sum;
    }
}

int main(void) { 

    return 0;
}
